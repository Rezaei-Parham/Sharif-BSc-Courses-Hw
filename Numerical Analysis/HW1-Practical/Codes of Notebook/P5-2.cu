
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
// sizes for shared memory
#define trsize 4
#define matsize 1024

// kernel for transposing matrix
__global__ void transpose(float *a, float* c){
    // row and column for the current thread
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    c[col*matsize + row] = a[row*matsize + col];
}

//CUDA kernel for multiplication
__global__ void multiply(const float *a, const float *b, float *c, int m){
    // defining row and column based on current thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < m && col < m){ // check for being inside the bounds
        float sum = 0;
        for (int i=0; i<m; i++){
            sum+= a[row*m+i] * b[i+m*col]; //dot product
        }
        c[row*m+col] = sum; //assign the value to the result
    }
}

void checkMultiplicationCorrect(float *p1, float *p2, float *p3, int m){
    for (int i=0; i<m; i++){
        for (int j=0; j<m; j++){
            float val = 0;
            for (int k=0; k<m; k++){
                val += p1[i*m+k] * p2[k*m+j];
            }
            // check for being close enough
            if (abs(p3[i*m+j] - val) > 0.0001){
                printf("\nwrong multiplication\n");
                return;
            }
        }
    }
    printf("\ncorrect multiplication\n");
}

int main(){
    int m = matsize;
    size_t size = m * m * sizeof(float);

    //allocate memory for host
    float *p1 = (float *)malloc(size);
    float *p2 = (float *)malloc(size);
    float *p3 = (float *)malloc(size);

    //initialize host memory
    for (int i=0; i<m*m; i++){
        p1[i] = (float)rand()/RAND_MAX;
        p2[i] = (float)rand()/RAND_MAX;
    }

    float *dp1, *dp2, *dp3, *dp4;
    hipMalloc(&dp1, size);
    hipMalloc(&dp2, size);
    hipMalloc(&dp3, size);
    hipMalloc(&dp4, size);

    // copy data to device from host
    hipMemcpy(dp1, p1, size, hipMemcpyHostToDevice);
    hipMemcpy(dp2, p2, size, hipMemcpyHostToDevice);

    //launch kernel
    int thr = trsize;
    int blc = m / thr;
    dim3 threads = dim3(thr, thr);
    dim3 blocks = dim3(blc, blc);
    // get the time of multiplication
    hipEvent_t begin, end;
    hipEventCreate(&begin);
    hipEventCreate(&end);
    hipEventRecord(begin);
    transpose<<<blocks, threads>>>(dp2, dp4);
    multiply<<<blocks, threads>>>(dp1, dp4, dp3, m);
    hipEventRecord(end);
    hipEventSynchronize(end);
    float time = 0;
    hipEventElapsedTime(&time, begin, end);
    // copy back
    hipMemcpy(p3, dp3, size, hipMemcpyDeviceToHost);

    printf("Matrix Multiplication size %d time %f\n",m, time/1000);
    checkMultiplicationCorrect(p1, p2, p3, m);
    //free memory
    hipFree(dp1);
    hipFree(dp2);
    hipFree(dp3);
    free(p1);
    free(p2);
    free(p3);
}
